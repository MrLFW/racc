#include <stdio.h>
#include <hip/hip_runtime.h>
//multiplication of a MxN matrix by a NxP matrix
#define N 3
#define M 2
#define P 2
#define BLOCK_SIZE 16

__global__ void matrixMultiply(int *a, int *b, int *c) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    

    if (row < M && col < P) {
        int sum = 0;
        for (int k = 0; k < N; k++) {
            sum += a[row * N + k] * b[k * P + col];
        }
        c[row * P + col] = sum;
    }
}

int main() {
    int a[M][N] = {{1, 2, 3}, {4, 5, 6}};
    int b[N][P] = {{1, 0 }, { 1, 0}, {1, 1}};
    int c[M][P] = {0};

    int *dev_a, *dev_b, *dev_c;
    hipMalloc((void **)&dev_a, M * N * sizeof(int));
    hipMalloc((void **)&dev_b, N * P * sizeof(int));
    hipMalloc((void **)&dev_c, M * P * sizeof(int));

    hipMemcpy(dev_a, a, M * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * P * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks( (P + threadsPerBlock.x - 1)/threadsPerBlock.x, (M + threadsPerBlock.y - 1)/threadsPerBlock.y );

    matrixMultiply<<<numBlocks, threadsPerBlock>>>(dev_a, dev_b, dev_c);


    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
       printf ("Kernel launch error: %s\n", hipGetErrorString(err));
       return 1;
    }


 
    hipMemcpy(c, dev_c, M * P * sizeof(int), hipMemcpyDeviceToHost);

    
    printf("Result:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < P; j++) {
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}


